#include "hip/hip_runtime.h"
#include<iostream>
#include<cub/block/block_reduce.cuh>

#define checkCUDA(expression)                              \
{                                                          \
  hipError_t status = (expression);                       \
  if (status != hipSuccess) {                             \
    std::cerr << "Error on line " << __LINE__ << ": "      \
              << hipGetErrorString(status) << std::endl;  \
    std::exit(EXIT_FAILURE);                               \
  }                                                        \
}

const int kBlockSize = 256;
const int kThreadElements = 4;

template<typename T>
void Print2D(const T* x, int N, int D, std::string msg) {
  printf("%s\n", msg.c_str());
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < D; j++) {
      printf("%f, ", static_cast<float>(x[j + i * D]));
    }
    printf("\n");
  }
}

template<typename T>
void Print1D(const T* x, int N, std::string msg) {
  printf("%s\n", msg.c_str());
  for (int i = 0; i < N; i++) {
    printf("%f, ", static_cast<float>(x[i]));
  }
  printf("\n");
}

template<typename T>
void IsClose2D(const T* x, const T* y, int N, int D, std::string msg) {
  bool is_same = true;
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < D; j++) {
      float d_val = static_cast<float>(x[j + i * D]);
      float h_val = static_cast<float>(y[j + i * D]);
      if (abs(d_val - h_val > 0.03f)) {
        is_same = false;
        printf("Found diff: CPU=%f, GPU=%f at (%d, %d)\n", h_val, d_val, i, j);
        break;
      }
    }
    if (!is_same) break;
  }
  printf("Test (%s): %s\n", msg.c_str(), is_same ? "True" : "False");
}

template<typename T>
void IsClose1D(const T* x, const T* y, int N, std::string msg) {
  bool is_same = true;
  for (int i = 0; i < N; i++) {
    float d_val = static_cast<float>(x[i]);
    float h_val = static_cast<float>(y[i]);
    if (abs(d_val - h_val > 0.03f)) {
      is_same = false;
      printf("Found diff: CPU=%f, GPU=%f at (%d,)\n", h_val, d_val, i);
      break;
    }
    if (!is_same) break;
  }
  printf("Test (%s): %s\n", msg.c_str(), is_same ? "True" : "False");
}

template<typename T, typename U>
__host__ __device__ U GetAs(const T* __restrict__ in, int offset) {
  return static_cast<U>(in[offset]);
}

template<typename T, typename U>
__device__ void GetStats(const T* __restrict__ row, const U epsilon,
                         U &mean, U &ivar, int tid, int D) {
  U sum = 0;
  for (int i = 0; i < D; i++) {
    sum += GetAs<T, U>(row, i);
  }
  mean = sum / D;
  U sum_ivar = 0;
  for (int i = 0; i < D; i++) {
    U curr = GetAs<T, U>(row, i);
    sum_ivar += (curr - mean) * (curr - mean);
  }
  ivar = rsqrt(sum_ivar / D + epsilon);
}

template<typename T, typename U>
__device__ void GetStatsV2(const T* __restrict__ row, const U epsilon,
                           U &mean, U &ivar, int tid, int D) {
  typedef hipcub::BlockReduce<U, kBlockSize> BlockReduce;
  __shared__ union {
      typename BlockReduce::TempStorage reduce;
      U broadcast[1];
  } temp_storage;
  U thread_data[kThreadElements];

  U sum = 0;
  const int workload_size = kBlockSize * kThreadElements;
  const int rounds = (D + workload_size - 1) / workload_size;
  int i = tid;
  for (int round = 0; round < rounds; round++) {
    for (int j = 0; j < kThreadElements; j++) {
      if (i * kThreadElements + j < D) {
        thread_data[j] = GetAs<T, U>(row, i * kThreadElements + j);
      } else {
        thread_data[j] = static_cast<U>(0);
      }
    }
    U aggregate = BlockReduce(temp_storage.reduce).Sum(thread_data);
    sum += aggregate;
    i += kBlockSize;
  }
  
  if (tid == 0) {
    temp_storage.broadcast[0] = sum;
  }
  __syncthreads();
  mean = temp_storage.broadcast[0] / D;

  U sum_ivar = 0;

  i = tid;
  for (int round = 0; round < rounds; round++) {
    for (int j = 0; j < kThreadElements; j++) {
      if (i * kThreadElements + j < D) {
        U curr = GetAs<T, U>(row, i * kThreadElements + j);
        thread_data[j] = (curr - mean) * (curr - mean);
      } else {
        thread_data[j] = static_cast<U>(0);
      }
    }
    U aggregate = BlockReduce(temp_storage.reduce).Sum(thread_data);
    sum_ivar += aggregate;
    i += kBlockSize;
  }
  
  if (tid == 0) {
    temp_storage.broadcast[0] = sum_ivar;
  }
  __syncthreads();
  ivar = rsqrt(temp_storage.broadcast[0] / D + epsilon);
}

// Like FusedBatchNormV3, we support T: {half, float} and U: {float}.
template<typename T, typename U>
__global__ void LayerNormKernel(const T* __restrict__ x,
                                const U* __restrict__ gamma,
                                const U* __restrict__ beta,
                                const U epsilon,
                                T* __restrict__ y,
                                U* __restrict__ cache_xivar,
                                U* __restrict__ cache_xmu,
                                int N, int D) {
  const int tid = threadIdx.x;
  const int row_stride = blockDim.x;
  const int col_stride = gridDim.x;

  for (int j = blockIdx.x; j < N; j += col_stride) {
    U mean, ivar;
    // GetStats(x + j * D, epsilon, mean, ivar, tid, D);
    GetStatsV2(x + j * D, epsilon, mean, ivar, tid, D);
    for (int i = tid; i < D; i += row_stride) {
      U curr = GetAs<T, U>(x, j * D + i);
      y[j * D + i] =
          static_cast<T>((curr - mean) * ivar * gamma[i] + beta[i]);
      // Intermediate results to speedup backprop.
      cache_xmu[j * D + i] = curr - mean;
    }
    cache_xivar[j] = ivar;
  }
}

template<typename T, typename U>
__global__ void LayerNormKernelV2Part1(const T* __restrict__ x,
                                       const U epsilon,
                                       U* __restrict__ cache_mean,
                                       U* __restrict__ cache_ivar,
                                       int N, int D) {
  const int tid = threadIdx.x;
  const int col_stride = gridDim.x;

  for (int j = blockIdx.x; j < N; j += col_stride) {
    U mean, ivar;
    // GetStats(x + j * D, epsilon, mean, ivar, tid, D);
    GetStatsV2(x + j * D, epsilon, mean, ivar, tid, D);
    // Intermediate results to speedup backprop.
    cache_ivar[j] = ivar;
    cache_mean[j] = mean;
  }
}

template<typename T, typename U>
__global__ void LayerNormKernelV2Part2(const T* __restrict__ x,
                                       const U* __restrict__ gamma,
                                       const U* __restrict__ beta,
                                       const U* __restrict__ cache_ivar,
                                       const U* __restrict__ cache_mean,
                                       const U epsilon,
                                       T* __restrict__ y,
                                       int N, int D) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid < N * D) {
    const int col = tid % D;
    const int row = tid / D;
    U mean = cache_mean[row];
    U ivar = cache_ivar[row];
    U curr = GetAs<T, U>(x, tid);
    y[tid] =
        static_cast<T>((curr - mean) * ivar * gamma[col] + beta[col]);
  }
}

template<typename T, typename U>
void LayerNormCPU(const T* x, const U* gamma, const U* beta, const U epsilon,
                  T* y, int N, int D) {
  for(int j = 0; j < N; j++) {
    U mean, ivar;
    U sum = 0;
    for(int i = 0; i < D; i++) {
      U curr = GetAs<T, U>(x, j * D + i);
      sum += curr;
    }
    mean = sum / D;
    U sum_ivar = 0;
    for (int i = 0; i < D; i++) {
      U curr = GetAs<T, U>(x, j * D + i);
      sum_ivar += (curr - mean) * (curr - mean);
    }
    ivar = rsqrt(sum_ivar / D + epsilon);

    for (int i = 0; i < D; i++) {
      U curr = GetAs<T, U>(x, j * D + i);
      y[j * D + i] =
          static_cast<T>((curr - mean) * ivar * gamma[i] + beta[i]);
    }
  }
}

template<typename T, typename U>
void LayerNormGradCPU(T* dy, T* x, U* cache_mean, U* cache_ivar, U* gamma, T* dx_h,
                      U* dgamma_h, U* dbeta_h, int N, int D) {
  // Compute dgamma, dbeta.
  for (int i = 0; i < D; i++) {
    dgamma_h[i] = 0;
    dbeta_h[i] = 0;
    for (int j = 0 ; j < N; j++) {
      U dy_curr = static_cast<U>(dy[j * D + i]);
      dgamma_h[i] += dy_curr * (x[j * D + i] - cache_mean[j]) * cache_ivar[j];
      dbeta_h[i] += dy_curr;
    }
  }

  // Compute dx.
  for (int i = 0; i < N; i++) {
    U dl_dvar = 0;
    for (int j = 0; j < D; j++) {
      U curr = static_cast<U>(dy[i * D + j]);
      dl_dvar += curr * gamma[j] * (x[i * D + j] - cache_mean[i]) * (-0.5) *
                     (cache_ivar[i] * cache_ivar[i] * cache_ivar[i]);
    }
    U dl_dmu = 0;
    for (int j = 0; j < D; j++) {
      U curr = static_cast<U>(dy[i * D + j]);
      dl_dmu += -1. * curr * gamma[j] * cache_ivar[i];
      dl_dmu += dl_dvar * (-2. / D) * (x[i * D + j] - cache_mean[i]);
    }

    for (int j = 0; j < D; j++) {
      U curr = static_cast<U>(dy[i * D + j]);
      U dl_di = curr * gamma[j] * cache_ivar[i];
      U di_dx = 1.;
      U dvar_dx = 2. * (x[i * D + j] - cache_mean[i]) / D;
      U dmu_dx = 1. / D;
      U dx = dl_di * di_dx + dl_dvar * dvar_dx + dl_dmu * dmu_dx;
      dx_h[i * D + j] = static_cast<T>(dx);
    }
  }

}

template<typename T, typename U>
__global__ void LayerNormGradBetaGamma(const T* __restrict__ dy,
                                       const U* __restrict__ cache_xmu,
                                       const U* __restrict__ cache_xivar,
                                       U* __restrict__ dgamma,
                                       U* __restrict__ dbeta, int N, int D) {
  const int tid = threadIdx.x;
  const int row_stride = blockDim.x;
  const int col_stride = gridDim.x;

  for (int j = blockIdx.x; j < N; j += col_stride) {
    for (int i = tid; i < D; i += row_stride) {
      U dy_curr = GetAs<T, U>(dy, j * D + i);
      atomicAdd(dgamma + i, dy_curr * cache_xmu[j * D + i] * cache_xivar[j]);
      atomicAdd(dbeta + i, dy_curr);
    }
  }
}

// To be replaced with "LaunchColumnReduction".
// TODO(kaixih): How to prepare the custom op.
template<typename T, typename U>
__global__ void LayerNormGradBetaGammaV2(const T* __restrict__ dy,
                                         const T* __restrict__ x,
                                         const U* __restrict__ cache_mean,
                                         const U* __restrict__ cache_ivar,
                                         U* __restrict__ dgamma,
                                         U* __restrict__ dbeta, int N, int D) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= D) return;

  U sum_dgamma = 0;
  U sum_dbeta = 0;
  for (int i = 0; i < N; i++) {
    U dy_curr = GetAs<T, U>(dy, i * D + tid);
    sum_dgamma += dy_curr * (x[i * D + tid] - cache_mean[i]) * cache_ivar[i];
    sum_dbeta += dy_curr;
  }

  dgamma[tid] = sum_dgamma;
  dbeta[tid] = sum_dbeta;
}

// To be replaced with "SetZero<T>".
template<typename U>
__global__ void InitGradBetaGamma(U* __restrict__ dgamma, U* __restrict__ dbeta,
                                  int D) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < D) {
    dgamma[tid] = 0;
    dbeta[tid] = 0;
  }
}

template<typename T, typename U>
__global__ void LayerNormGradInput(const T* __restrict__ dy,
                                   const T* __restrict__ x,
                                   const U* __restrict__ gamma,
                                   const U* __restrict__ cache_mean,
                                   const U* __restrict__ cache_ivar,
                                   T * dx, int N, int D) {
  const int tid = threadIdx.x;
  const int row_stride = blockDim.x;
  const int col_stride = gridDim.x;

  typedef hipcub::BlockReduce<U, kBlockSize> BlockReduce;
  __shared__ union {
      typename BlockReduce::TempStorage reduce;
      U broadcast[1];
  } temp_storage;
  U thread_data[kThreadElements];

  const int workload_size = kBlockSize * kThreadElements;
  const int rounds = (D + workload_size - 1) / workload_size;
  for (int k = blockIdx.x; k < N; k += col_stride) {
    int i = tid;
    U dl_dvar = 0;
    for (int round = 0; round < rounds; round++) {
      for (int j = 0; j < kThreadElements; j++) {
        int row_offset = i * kThreadElements + j;
        if (row_offset < D) {
          U curr = GetAs<T, U>(dy, k * D + row_offset);
          thread_data[j] = curr * gamma[row_offset] *
                           (x[k * D + row_offset] - cache_mean[k]) *
                           (-0.5) * (cache_ivar[k] * cache_ivar[k] *
                                     cache_ivar[k]);
        } else {
          thread_data[j] = static_cast<U>(0);
        }
      }
      U aggregate = BlockReduce(temp_storage.reduce).Sum(thread_data);
      dl_dvar += aggregate;
      i += kBlockSize;
    }
    
    if (tid == 0) {
      temp_storage.broadcast[0] = dl_dvar;
    }
    __syncthreads();
    dl_dvar = temp_storage.broadcast[0];

    i = tid;
    U dl_dmu = 0;
    for (int round = 0; round < rounds; round++) {
      for (int j = 0; j < kThreadElements; j++) {
        int row_offset = i * kThreadElements + j;
        if (row_offset < D) {
          U curr = GetAs<T, U>(dy, k * D + row_offset);
          thread_data[j] = -1. * curr * gamma[row_offset] *
                           cache_ivar[k] + dl_dvar * (-2. / D) *
                           (x[k * D + row_offset] - cache_mean[k]);
        } else {
          thread_data[j] = static_cast<U>(0);
        }
      }
      U aggregate = BlockReduce(temp_storage.reduce).Sum(thread_data);
      dl_dmu += aggregate;
      i += kBlockSize;
    }
    
    if (tid == 0) {
      temp_storage.broadcast[0] = dl_dmu;
    }
    __syncthreads();
    dl_dmu = temp_storage.broadcast[0];

    for (int i = tid; i < D; i += row_stride) {
      U curr = GetAs<T, U>(dy, k * D + i);
      U dl_di = curr * gamma[i] * cache_ivar[k];
      U di_dx = 1.;
      U dvar_dx = 2. * (x[k * D + i] - cache_mean[k]) / D;
      U dmu_dx = 1. / D;
      U dl_dx = dl_di * di_dx + dl_dvar * dvar_dx + dl_dmu * dmu_dx;
      dx[k * D + i] = static_cast<T>(dl_dx);
    }
  }
}

template<typename T, typename U>
void LayerNormGradGPU(T* dy, T* x, U* cache_mean, U* cache_ivar, U* gamma,
                      T* dx, U* dgamma, U* dbeta, int N, int D) {

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  dim3 threads_init(128, 1, 1);
  dim3 blocks_init((D + 127) / 128, 1, 1);
  InitGradBetaGamma<<<blocks_init, threads_init>>>(dgamma, dbeta, D);

  // dim3 threads(kBlockSize, 1, 1);
  // dim3 blocks(N, 1, 1);
  // LayerNormGradBetaGamma<<<blocks, threads>>>(
      // dy, cache_xmu, cache_xivar, dgamma, dbeta, N, D);
  dim3 threads(kBlockSize, 1, 1);
  dim3 blocks((D + kBlockSize - 1) / kBlockSize, 1, 1);
  LayerNormGradBetaGammaV2<<<blocks, threads>>>(
      dy, x, cache_mean, cache_ivar, dgamma, dbeta, N, D);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU time (dgamma, dbeta): %f ms\n", milliseconds);

  hipEventRecord(start);
  dim3 threads_input(kBlockSize, 1, 1);
  dim3 blocks_input(N, 1, 1);
  LayerNormGradInput<<<blocks_input, threads_input>>>(
      dy, x, gamma, cache_mean, cache_ivar, dx, N, D);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU time (dx): %f ms\n", milliseconds);

}

template<typename T>
void PrepareAlloc(T **x, int size, bool use_host, bool human_readable,
                  int seed=99, int init=-1) {
  srand(seed);
  int max_int = 32768;
  T *buf_x = new T[size];
  for (int i = 0; i < size; i++) {
    if (init != -1) {
      buf_x[i] = init;
    } else if (human_readable) {
      buf_x[i] = i / 10.;
    } else {
      buf_x[i] = static_cast<T>(static_cast<float>(rand() % max_int) / max_int);
    }
  }
  if (use_host) {
    checkCUDA(hipMallocManaged(&(*x), size * sizeof(T)));
    for (int i = 0; i < size; i++) {
      (*x)[i] = buf_x[i];
    }
  } else {
    checkCUDA(hipMalloc(&(*x), size * sizeof(T)));
    checkCUDA(hipMemcpy(*x, buf_x,  size * sizeof(T), hipMemcpyHostToDevice));
  }

  delete[] buf_x;
}

#define DTYPE float

int main() {

  /** Parameters and Knobs **/
  int N = 10;
  int D = 10000000;
  bool allow_print = false;
  bool human_readable = false;
  bool use_host = false;

  DTYPE* x;
  float* gamma;
  float* beta;
  DTYPE* y;
  float* cache_xivar;
  float* cache_xmu;
  float* cache_ivar;
  float* cache_mean;

  PrepareAlloc(&x, N * D, use_host, human_readable, 12);
  PrepareAlloc(&gamma, D, use_host, human_readable, 13);
  PrepareAlloc(&beta, D, use_host, human_readable, 14);
  PrepareAlloc(&y, N * D, use_host, human_readable);

  PrepareAlloc(&cache_xivar, N, use_host, human_readable);
  PrepareAlloc(&cache_xmu, N * D, use_host, human_readable);

  PrepareAlloc(&cache_ivar, N, use_host, human_readable);
  PrepareAlloc(&cache_mean, N, use_host, human_readable);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  // const dim3 threads(kBlockSize, 1);
  // const dim3 blocks(N, 1, 1);
  // LayerNormKernel<<<blocks, threads>>>(x, gamma, beta, 0.001f, y, cache_xivar,
                                       // cache_xmu, N, D);
  const dim3 threads(kBlockSize, 1);
  const dim3 blocks(N, 1, 1);
  LayerNormKernelV2Part1<<<blocks, threads>>>(x, 0.001f, cache_ivar, cache_mean, N, D);
  const dim3 threads_x(kBlockSize, 1);
  const dim3 blocks_x((N * D + kBlockSize - 1) / kBlockSize, 1, 1);
  LayerNormKernelV2Part2<<<blocks_x, threads_x>>>(x, gamma, beta, cache_ivar,
                                                  cache_mean, 0.001f, y, N, D);

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU time (y): %f ms\n", milliseconds);

  checkCUDA(hipDeviceSynchronize());
  if (use_host && allow_print) {
    Print2D(y, N, D, "GPU y:");
  }

  if (use_host) {
    DTYPE *y_h = new DTYPE[N * D];
    LayerNormCPU(x, gamma, beta, 0.001f, y_h, N, D);
    if (allow_print) {
      Print2D(y_h, N, D, "CPU y:");
    }

    IsClose2D(y, y_h, N, D, "y");
    delete[] y_h;
  }

  DTYPE* dy;
  DTYPE* dx;
  float* dgamma;
  float* dbeta;

  PrepareAlloc(&dy, N * D, use_host, human_readable, 99, 1);
  PrepareAlloc(&dx, N * D, use_host, human_readable);
  PrepareAlloc(&dgamma, D, use_host, human_readable);
  PrepareAlloc(&dbeta, D, use_host, human_readable);

  LayerNormGradGPU(dy, x, cache_mean, cache_ivar, gamma, dx, dgamma, dbeta, N, D);
  checkCUDA(hipDeviceSynchronize());
  if (use_host && allow_print) {
    Print1D(dgamma, D, "GPU dgamma:");
    Print1D(dbeta, D, "GPU dbeta:");
    Print2D(dx, N, D, "GPU dx:");
  }

  if (use_host) {
    DTYPE *dx_h = new DTYPE[N * D];
    float *dgamma_h = new float[D];
    float *dbeta_h = new float[D];
    LayerNormGradCPU(
        dy, x, cache_mean, cache_ivar, gamma, dx_h, dgamma_h, dbeta_h, N, D);
    if (allow_print) {
      Print1D(dgamma_h, D, "CPU dgamma:");
      Print1D(dbeta_h, D, "CPU dbeta:");
      Print2D(dx_h, N, D, "CPU dx:");
    }

    IsClose1D(dgamma, dgamma_h, D, "dgamma");
    IsClose1D(dbeta, dbeta_h, D, "dbeta");
    IsClose2D(dx, dx_h, N, D, "dx");

    delete[] dx_h;
    delete[] dgamma_h;
    delete[] dbeta_h;
  }

  checkCUDA(hipFree(x));
  checkCUDA(hipFree(gamma));
  checkCUDA(hipFree(beta));
  checkCUDA(hipFree(y));
  checkCUDA(hipFree(dy));
  checkCUDA(hipFree(dx));
  checkCUDA(hipFree(dgamma));
  checkCUDA(hipFree(dbeta));
}
